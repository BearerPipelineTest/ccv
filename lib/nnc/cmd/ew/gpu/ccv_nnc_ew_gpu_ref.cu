#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_reciprocal_kernel(const size_t count, const NUM1* const a, NUM2* const b)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		b[i] = (NUM2)((NUM1)1. / a[i]);
	}
}

template<typename NUM1, typename NUM2, typename NUM3>
__global__ void _ccv_nnc_ewdiv_kernel(const size_t count, const NUM1* const a, const NUM2* const b, NUM3* const c)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		c[i] = (NUM3)(a[i] / (NUM1)b[i]);
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_reciprocal_kernel(const size_t count, const float p, const NUM1* const a, NUM2* const b)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		b[i] = (NUM2)((NUM1)p / a[i]);
	}
}

template<typename NUM1, typename NUM2, typename NUM3>
__global__ void _ccv_nnc_ewdiv_kernel(const size_t count, const float p, const NUM1* const a, const NUM2* const b, NUM3* const c)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		c[i] = (NUM3)((NUM1)p * a[i] / (NUM1)b[i]);
	}
}

static int _ccv_nnc_ewdiv_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 2);
	const ccv_nnc_tensor_t* const a = inputs[0];
	const ccv_nnc_tensor_t* const b = inputs[1];
	assert(CCV_IS_TENSOR_CONTIGUOUS(b));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const c = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(c));
	const size_t count = ccv_nnc_tensor_count(b->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && b->info.dim[i] > 0; i++)
		{ assert(b->info.dim[i] == c->info.dim[i]); }
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	if (a)
	{
		assert(CCV_IS_TENSOR_CONTIGUOUS(a));
		assert(a->info.datatype == b->info.datatype);
		for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
			{ assert(a->info.dim[i] == b->info.dim[i]); }
		if (a->info.datatype == CCV_32F && c->info.datatype == CCV_32F)
		{
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, b->data.f32, c->data.f32);
		} else if (a->info.datatype == CCV_32F && c->info.datatype == CCV_16F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, b->data.f32, (__half*)c->data.f16);
		} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_32F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)b->data.f16, c->data.f32);
		} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_16F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)b->data.f16, (__half*)c->data.f16);
		}
	} else {
		if (b->info.datatype == CCV_32F && c->info.datatype == CCV_32F)
		{
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, c->data.f32);
		} else if (b->info.datatype == CCV_32F && c->info.datatype == CCV_16F) {
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, (__half*)c->data.f16);
		} else if (b->info.datatype == CCV_16F && c->info.datatype == CCV_32F) {
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, c->data.f32);
		} else if (b->info.datatype == CCV_16F && c->info.datatype == CCV_16F) {
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)c->data.f16);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_ewnegmuldiv_kernel(const size_t count, const NUM1* const g, const NUM2* const b, const NUM1* const c, NUM2* const h)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		h[i] = (NUM2)(-g[i] * c[i] / (NUM1)b[i]);
	}
}

static int _ccv_nnc_ewdiv_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	const ccv_nnc_tensor_t* const g = inputs[0]; // gradient
	assert(CCV_IS_TENSOR_CONTIGUOUS(g));
	const size_t count = ccv_nnc_tensor_count(g->info);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	if (outputs[0])
	{
		const ccv_nnc_tensor_t* const b = inputs[2];
		assert(CCV_IS_TENSOR_CONTIGUOUS(b));
		ccv_nnc_tensor_t* const h = outputs[0];
		assert(CCV_IS_TENSOR_CONTIGUOUS(h));
		assert(ccv_nnc_tensor_count(h->info) == count);
		assert(b->info.datatype == h->info.datatype);
		if (b->info.datatype == CCV_32F && g->info.datatype == CCV_32F)
		{
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, g->data.f32, b->data.f32, h->data.f32);
		} else if (b->info.datatype == CCV_32F && g->info.datatype == CCV_16F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)g->data.f16, b->data.f32, h->data.f32);
		} else if (b->info.datatype == CCV_16F && g->info.datatype == CCV_32F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, g->data.f32, (__half*)b->data.f16, (__half*)h->data.f16);
		} else if (b->info.datatype == CCV_16F && g->info.datatype == CCV_16F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)g->data.f16, (__half*)b->data.f16, (__half*)h->data.f16);
		}
	}
	if (output_size >= 2 && outputs[1])
	{
		const ccv_nnc_tensor_t* const b = inputs[2];
		assert(CCV_IS_TENSOR_CONTIGUOUS(b));
		const ccv_nnc_tensor_t* const c = inputs[3];
		assert(CCV_IS_TENSOR_CONTIGUOUS(c));
		ccv_nnc_tensor_t* const h = outputs[1];
		assert(CCV_IS_TENSOR_CONTIGUOUS(h));
		assert(ccv_nnc_tensor_count(h->info) == count);
		assert(b->info.datatype == h->info.datatype);
		assert(c->info.datatype == g->info.datatype);
		if (b->info.datatype == CCV_32F && g->info.datatype == CCV_32F)
		{
			_ccv_nnc_ewnegmuldiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, g->data.f32, b->data.f32, c->data.f32, h->data.f32);
		} else if (b->info.datatype == CCV_32F && g->info.datatype == CCV_16F) {
			_ccv_nnc_ewnegmuldiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)g->data.f16, b->data.f32, (__half*)c->data.f16, h->data.f32);
		} else if (b->info.datatype == CCV_16F && g->info.datatype == CCV_32F) {
			_ccv_nnc_ewnegmuldiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, g->data.f32, (__half*)b->data.f16, c->data.f32, (__half*)h->data.f16);
		} else if (b->info.datatype == CCV_16F && g->info.datatype == CCV_16F) {
			_ccv_nnc_ewnegmuldiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)g->data.f16, (__half*)b->data.f16, (__half*)c->data.f16, (__half*)h->data.f16);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_EWDIV_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_ewdiv_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_EWDIV_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_ewdiv_back;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_ewexp_kernel(const size_t count, const NUM1* const a, NUM2* const c)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		c[i] = (NUM2)exp((float)a[i]);
	}
}

static int _ccv_nnc_ewexp_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 1);
	const ccv_nnc_tensor_t* const a = inputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(a));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const c = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(c));
	const size_t count = ccv_nnc_tensor_count(a->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
		{ assert(a->info.dim[i] == c->info.dim[i]); }
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	if (a->info.datatype == CCV_32F && c->info.datatype == CCV_32F)
	{
		_ccv_nnc_ewexp_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, c->data.f32);
	} else if (a->info.datatype == CCV_32F && c->info.datatype == CCV_16F) {
		_ccv_nnc_ewexp_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, (__half*)c->data.f16);
	} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_32F) {
		_ccv_nnc_ewexp_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, c->data.f32);
	} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_16F) {
		_ccv_nnc_ewexp_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)c->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_copy_kernel(const size_t count, const NUM1* const a, NUM2* const c)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		c[i] = (NUM2)a[i];
	}
}

template<typename NUM1, typename NUM2, typename NUM3>
__global__ void _ccv_nnc_ewmul_kernel(const size_t count, const NUM1* const a, const NUM2* const b, NUM3* const c)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		c[i] = (NUM3)(a[i] * (NUM1)b[i]);
	}
}

static int _ccv_nnc_ewexp_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const ccv_nnc_tensor_t* const b = inputs[2];
	assert(CCV_IS_TENSOR_CONTIGUOUS(b));
	ccv_nnc_tensor_t* const h = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(h));
	assert(b->info.datatype == h->info.datatype);
	if (inputs[0] == 0)
	{
		const size_t count = ccv_nnc_tensor_count(b->info);
		assert(ccv_nnc_tensor_count(h->info) == count);
		if (b->info.datatype == CCV_32F && h->info.datatype == CCV_32F)
		{
			_ccv_nnc_copy_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, h->data.f32);
		} else if (b->info.datatype == CCV_32F && h->info.datatype == CCV_16F) {
			_ccv_nnc_copy_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, h->data.f32);
		} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_32F) {
			_ccv_nnc_copy_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, (__half*)h->data.f16);
		} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_16F) {
			_ccv_nnc_copy_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)h->data.f16);
		}
	} else {
		const ccv_nnc_tensor_t* const g = inputs[0]; // gradient
		assert(CCV_IS_TENSOR_CONTIGUOUS(g));
		const size_t count = ccv_nnc_tensor_count(g->info);
		assert(ccv_nnc_tensor_count(h->info) == count);
		if (b->info.datatype == CCV_32F && g->info.datatype == CCV_32F)
		{
			_ccv_nnc_ewmul_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, g->data.f32, b->data.f32, h->data.f32);
		} else if (b->info.datatype == CCV_32F && g->info.datatype == CCV_16F) {
			_ccv_nnc_ewmul_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)g->data.f16, b->data.f32, h->data.f32);
		} else if (b->info.datatype == CCV_16F && g->info.datatype == CCV_32F) {
			_ccv_nnc_ewmul_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, g->data.f32, (__half*)b->data.f16, (__half*)h->data.f16);
		} else if (b->info.datatype == CCV_16F && g->info.datatype == CCV_16F) {
			_ccv_nnc_ewmul_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)g->data.f16, (__half*)b->data.f16, (__half*)h->data.f16);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_EWEXP_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_ewexp_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_EWEXP_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_ewexp_back;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_ewlog_kernel(const size_t count, const NUM1* const a, NUM2* const c)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		c[i] = (NUM2)log((float)a[i]);
	}
}

static int _ccv_nnc_ewlog_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 1);
	const ccv_nnc_tensor_t* const a = inputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(a));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const c = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(c));
	const size_t count = ccv_nnc_tensor_count(a->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
		{ assert(a->info.dim[i] == c->info.dim[i]); }
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	if (a->info.datatype == CCV_32F && c->info.datatype == CCV_32F)
	{
		_ccv_nnc_ewlog_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, c->data.f32);
	} else if (a->info.datatype == CCV_32F && c->info.datatype == CCV_16F) {
		_ccv_nnc_ewlog_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, (__half*)c->data.f16);
	} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_32F) {
		_ccv_nnc_ewlog_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, c->data.f32);
	} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_16F) {
		_ccv_nnc_ewlog_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)c->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_ewlog_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	const ccv_nnc_tensor_t* const a = inputs[0];
	const ccv_nnc_tensor_t* const b = inputs[1];
	assert(CCV_IS_TENSOR_CONTIGUOUS(b));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const c = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(c));
	const size_t count = ccv_nnc_tensor_count(b->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && b->info.dim[i] > 0; i++)
		{ assert(b->info.dim[i] == c->info.dim[i]); }
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	if (a)
	{
		assert(CCV_IS_TENSOR_CONTIGUOUS(a));
		assert(a->info.datatype == b->info.datatype);
		for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
			{ assert(a->info.dim[i] == b->info.dim[i]); }
		if (a->info.datatype == CCV_32F && c->info.datatype == CCV_32F)
		{
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, b->data.f32, c->data.f32);
		} else if (a->info.datatype == CCV_32F && c->info.datatype == CCV_16F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, b->data.f32, (__half*)c->data.f16);
		} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_32F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)b->data.f16, c->data.f32);
		} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_16F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)b->data.f16, (__half*)c->data.f16);
		}
	} else {
		if (b->info.datatype == CCV_32F && c->info.datatype == CCV_32F)
		{
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, c->data.f32);
		} else if (b->info.datatype == CCV_32F && c->info.datatype == CCV_16F) {
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, (__half*)c->data.f16);
		} else if (b->info.datatype == CCV_16F && c->info.datatype == CCV_32F) {
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, c->data.f32);
		} else if (b->info.datatype == CCV_16F && c->info.datatype == CCV_16F) {
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)c->data.f16);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_EWLOG_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_ewlog_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_EWLOG_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_ewlog_back;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_ewsqrt_kernel(const size_t count, const NUM1* const a, NUM2* const c)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		c[i] = (NUM2)sqrt((float)a[i]);
	}
}

static int _ccv_nnc_ewsqrt_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 1);
	const ccv_nnc_tensor_t* const a = inputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(a));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const c = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(c));
	const size_t count = ccv_nnc_tensor_count(a->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
		{ assert(a->info.dim[i] == c->info.dim[i]); }
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	if (a->info.datatype == CCV_32F && c->info.datatype == CCV_32F)
	{
		_ccv_nnc_ewsqrt_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, c->data.f32);
	} else if (a->info.datatype == CCV_32F && c->info.datatype == CCV_16F) {
		_ccv_nnc_ewsqrt_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, (__half*)c->data.f16);
	} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_32F) {
		_ccv_nnc_ewsqrt_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, c->data.f32);
	} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_16F) {
		_ccv_nnc_ewsqrt_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)c->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_ewsqrt_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	const ccv_nnc_tensor_t* const a = inputs[0];
	const ccv_nnc_tensor_t* const b = inputs[2];
	assert(CCV_IS_TENSOR_CONTIGUOUS(b));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const c = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(c));
	const size_t count = ccv_nnc_tensor_count(b->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && b->info.dim[i] > 0; i++)
		{ assert(b->info.dim[i] == c->info.dim[i]); }
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	if (a)
	{
		assert(CCV_IS_TENSOR_CONTIGUOUS(a));
		assert(a->info.datatype == b->info.datatype);
		for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
			{ assert(a->info.dim[i] == b->info.dim[i]); }
		if (a->info.datatype == CCV_32F && c->info.datatype == CCV_32F)
		{
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, 0.5, a->data.f32, b->data.f32, c->data.f32);
		} else if (a->info.datatype == CCV_32F && c->info.datatype == CCV_16F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, 0.5, a->data.f32, b->data.f32, (__half*)c->data.f16);
		} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_32F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, 0.5, (__half*)a->data.f16, (__half*)b->data.f16, c->data.f32);
		} else if (a->info.datatype == CCV_16F && c->info.datatype == CCV_16F) {
			_ccv_nnc_ewdiv_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, 0.5, (__half*)a->data.f16, (__half*)b->data.f16, (__half*)c->data.f16);
		}
	} else {
		if (b->info.datatype == CCV_32F && c->info.datatype == CCV_32F)
		{
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, 0.5, b->data.f32, c->data.f32);
		} else if (b->info.datatype == CCV_32F && c->info.datatype == CCV_16F) {
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, 0.5, b->data.f32, (__half*)c->data.f16);
		} else if (b->info.datatype == CCV_16F && c->info.datatype == CCV_32F) {
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, 0.5, (__half*)b->data.f16, c->data.f32);
		} else if (b->info.datatype == CCV_16F && c->info.datatype == CCV_16F) {
			_ccv_nnc_reciprocal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, 0.5, (__half*)b->data.f16, (__half*)c->data.f16);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_EWSQRT_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_ewsqrt_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_EWSQRT_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_ewsqrt_back;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_clamp_kernel(const size_t count, const NUM1* const a, NUM2* const b, const float minv, const float maxv)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		b[i] = (NUM2)min(max(a[i], minv), maxv);
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_min_kernel(const size_t count, const NUM1* const a, NUM2* const b, const float maxv)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		b[i] = (NUM2)min(a[i], maxv);
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_max_kernel(const size_t count, const NUM1* const a, NUM2* const b, const float minv)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		b[i] = (NUM2)max(a[i], minv);
	}
}

static int _ccv_nnc_clamp_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 1);
	const ccv_nnc_tensor_t* const a = inputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(a));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const b = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(b));
	const size_t count = ccv_nnc_tensor_count(a->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
		{ assert(a->info.dim[i] == b->info.dim[i]); }
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const float minv = cmd.info.clamp.min;
	const float maxv = cmd.info.clamp.max;
	assert(!isnan(minv) || !isnan(maxv));
	if (isnan(minv))
	{
		if (a->info.datatype == CCV_32F && b->info.datatype == CCV_32F)
		{
			_ccv_nnc_min_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, b->data.f32, maxv);
		} else if (a->info.datatype == CCV_32F && b->info.datatype == CCV_16F) {
			_ccv_nnc_min_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, (__half*)b->data.f16, maxv);
		} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_32F) {
			_ccv_nnc_min_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, b->data.f32, maxv);
		} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_16F) {
			_ccv_nnc_min_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)b->data.f16, maxv);
		}
	} else if (isnan(maxv)) {
		if (a->info.datatype == CCV_32F && b->info.datatype == CCV_32F)
		{
			_ccv_nnc_max_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, b->data.f32, minv);
		} else if (a->info.datatype == CCV_32F && b->info.datatype == CCV_16F) {
			_ccv_nnc_max_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, (__half*)b->data.f16, minv);
		} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_32F) {
			_ccv_nnc_max_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, b->data.f32, minv);
		} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_16F) {
			_ccv_nnc_max_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)b->data.f16, minv);
		}
	} else {
		if (a->info.datatype == CCV_32F && b->info.datatype == CCV_32F)
		{
			_ccv_nnc_clamp_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, b->data.f32, minv, maxv);
		} else if (a->info.datatype == CCV_32F && b->info.datatype == CCV_16F) {
			_ccv_nnc_clamp_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, a->data.f32, (__half*)b->data.f16, minv, maxv);
		} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_32F) {
			_ccv_nnc_clamp_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, b->data.f32, minv, maxv);
		} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_16F) {
			_ccv_nnc_clamp_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)a->data.f16, (__half*)b->data.f16, minv, maxv);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_clamp_kernel_back(const size_t count, const NUM1* const b, const NUM1* const g, NUM2* const h, const NUM1 minv, const NUM1 maxv)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		h[i] = (NUM2)((b[i] <= minv || b[i] >= maxv) ? (NUM1)0.0 : g[i]);
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_clamp_kernel_back(const size_t count, const NUM1* const b, NUM2* const h, const NUM1 minv, const NUM1 maxv)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		h[i] = (NUM2)((b[i] <= minv || b[i] >= maxv) ? 0.0 : 1.0);
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_min_kernel_back(const size_t count, const NUM1* const b, const NUM1* const g, NUM2* const h, const NUM1 maxv)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		h[i] = (NUM2)((b[i] >= maxv) ? (NUM1)0.0 : g[i]);
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_min_kernel_back(const size_t count, const NUM1* const b, NUM2* const h, const NUM1 maxv)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		h[i] = (NUM2)((b[i] >= maxv) ? 0.0 : 1.0);
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_max_kernel_back(const size_t count, const NUM1* const b, const NUM1* const g, NUM2* const h, const NUM1 minv)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		h[i] = (NUM2)((b[i] <= minv) ? (NUM1)0.0 : g[i]);
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_max_kernel_back(const size_t count, const NUM1* const b, NUM2* const h, const NUM1 minv)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		h[i] = (NUM2)((b[i] <= minv) ? 0.0 : 1.0);
	}
}

static int _ccv_nnc_clamp_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 3);
	const ccv_nnc_tensor_t* const g = inputs[0];
	assert(!g || !CCV_IS_TENSOR_VIEW(g));
	const ccv_nnc_tensor_t* const b = inputs[2];
	assert(CCV_IS_TENSOR_CONTIGUOUS(b));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const h = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(h));
	const size_t count = ccv_nnc_tensor_count(b->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && b->info.dim[i] > 0; i++)
		{ assert(b->info.dim[i] == h->info.dim[i]); }
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const float minv = cmd.info.clamp.min;
	const float maxv = cmd.info.clamp.max;
	assert(!isnan(minv) || !isnan(maxv));
	if (g)
	{
		for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && b->info.dim[i] > 0; i++)
			{ assert(b->info.dim[i] == g->info.dim[i]); }
		assert(g->info.datatype == b->info.datatype);
		if (isnan(minv))
		{
			if (b->info.datatype == CCV_32F && h->info.datatype == CCV_32F)
			{
				_ccv_nnc_min_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, g->data.f32, h->data.f32, maxv);
			} else if (b->info.datatype == CCV_32F && h->info.datatype == CCV_16F) {
				_ccv_nnc_min_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, g->data.f32, (__half*)h->data.f16, maxv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_32F) {
				_ccv_nnc_min_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)g->data.f16, h->data.f32, (__half)maxv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_16F) {
				_ccv_nnc_min_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)g->data.f16, (__half*)h->data.f16, (__half)maxv);
			}
		} else if (isnan(maxv)) {
			if (b->info.datatype == CCV_32F && h->info.datatype == CCV_32F)
			{
				_ccv_nnc_max_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, g->data.f32, h->data.f32, minv);
			} else if (b->info.datatype == CCV_32F && h->info.datatype == CCV_16F) {
				_ccv_nnc_max_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, g->data.f32, (__half*)h->data.f16, minv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_32F) {
				_ccv_nnc_max_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)g->data.f16, h->data.f32, (__half)minv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_16F) {
				_ccv_nnc_max_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)g->data.f16, (__half*)h->data.f16, (__half)minv);
			}
		} else {
			if (b->info.datatype == CCV_32F && h->info.datatype == CCV_32F)
			{
				_ccv_nnc_clamp_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, g->data.f32, h->data.f32, minv, maxv);
			} else if (b->info.datatype == CCV_32F && h->info.datatype == CCV_16F) {
				_ccv_nnc_clamp_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, g->data.f32, (__half*)h->data.f16, minv, maxv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_32F) {
				_ccv_nnc_clamp_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)g->data.f16, h->data.f32, (__half)minv, (__half)maxv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_16F) {
				_ccv_nnc_clamp_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)g->data.f16, (__half*)h->data.f16, (__half)minv, (__half)maxv);
			}
		}
	} else {
		if (isnan(minv))
		{
			if (b->info.datatype == CCV_32F && h->info.datatype == CCV_32F)
			{
				_ccv_nnc_min_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, h->data.f32, maxv);
			} else if (b->info.datatype == CCV_32F && h->info.datatype == CCV_16F) {
				_ccv_nnc_min_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, (__half*)h->data.f16, maxv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_32F) {
				_ccv_nnc_min_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, h->data.f32, (__half)maxv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_16F) {
				_ccv_nnc_min_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)h->data.f16, (__half)maxv);
			}
		} else if (isnan(maxv)) {
			if (b->info.datatype == CCV_32F && h->info.datatype == CCV_32F)
			{
				_ccv_nnc_max_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, h->data.f32, minv);
			} else if (b->info.datatype == CCV_32F && h->info.datatype == CCV_16F) {
				_ccv_nnc_max_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, (__half*)h->data.f16, minv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_32F) {
				_ccv_nnc_max_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, h->data.f32, (__half)minv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_16F) {
				_ccv_nnc_max_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)h->data.f16, (__half)minv);
			}
		} else {
			if (b->info.datatype == CCV_32F && h->info.datatype == CCV_32F)
			{
				_ccv_nnc_clamp_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, h->data.f32, minv, maxv);
			} else if (b->info.datatype == CCV_32F && h->info.datatype == CCV_16F) {
				_ccv_nnc_clamp_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, b->data.f32, (__half*)h->data.f16, minv, maxv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_32F) {
				_ccv_nnc_clamp_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, h->data.f32, (__half)minv, (__half)maxv);
			} else if (b->info.datatype == CCV_16F && h->info.datatype == CCV_16F) {
				_ccv_nnc_clamp_kernel_back<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, (__half*)b->data.f16, (__half*)h->data.f16, (__half)minv, (__half)maxv);
			}
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_CLAMP_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_clamp_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_CLAMP_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_clamp_back;
}
