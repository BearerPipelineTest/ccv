extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

static int _ccv_nnc_reduce_sum_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 1);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	ccv_nnc_tensor_view_t atv = ccv_nnc_get_tensor_view(inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	ccv_nnc_tensor_view_t btv = ccv_nnc_get_tensor_view(outputs[0]);
	ccv_nnc_tensor_view_t* tvs[] = {
		&atv, &btv
	};
	ccv_nnc_tensor_view_alignment(tvs, 2);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &btv);
	hipdnnReduceTensorDescriptor_t reduce_sum = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
	hipdnnSetReduceTensorDescriptor(reduce_sum, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	void* workspace = 0;
	size_t workspace_size = 0;
	CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce_sum, a.descriptor, b.descriptor, &workspace_size));
	if (workspace_size)
	{
		workspace = ccv_nnc_stream_context_get_workspace(stream_context, workspace_size, CCV_TENSOR_GPU_MEMORY);
		assert(workspace);
	}
	static const float one = 1, zero = 0;
	CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce_sum, 0, 0, workspace, workspace_size, &one, a.descriptor, a.data.u8, &zero, b.descriptor, b.data.u8));
	ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce_sum);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_reduce_sum_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	static const float one = 1;
	ccv_nnc_tensor_view_t* const atv = (ccv_nnc_tensor_view_t*)outputs[0];
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, atv);
	if (inputs[0] == 0)
	{
		CUDNN_ENFORCE(hipdnnSetTensor(cudnn, a.descriptor, a.data.u8, &one));
	} else {
		static const float zero = 0;
		ccv_nnc_tensor_view_t gtv = ccv_nnc_get_tensor_view(inputs[0]);
		ccv_nnc_tensor_view_t* tvs[] = {
			atv, &gtv
		};
		ccv_nnc_tensor_view_alignment(tvs, 2);
		const ccv_nnc_cudnn_tensor_view_descriptor_t g = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &gtv);
		CUDNN_ENFORCE(hipdnnAddTensor(cudnn, &one, g.descriptor, g.data.u8, &zero, a.descriptor, a.data.u8));
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(g);
	}
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_REDUCE_SUM_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_reduce_sum_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_REDUCE_SUM_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_reduce_sum_back;
#endif
}

