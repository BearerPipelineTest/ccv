extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

static int _ccv_nnc_reduce_mean_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 1);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	ccv_nnc_tensor_view_t atv = ccv_nnc_get_tensor_view(inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	ccv_nnc_tensor_view_t btv = ccv_nnc_get_tensor_view(outputs[0]);
	ccv_nnc_tensor_view_t* tvs[] = {
		&atv, &btv
	};
	ccv_nnc_tensor_view_alignment(tvs, 2);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &btv);
	int can_reduce = 0;
	int i;
	for (i = 0; !can_reduce && i < cmd.info.reduce.count; i++)
		can_reduce = (inputs[0]->info.dim[cmd.info.reduce.axis[i]] > 1);
	static const float one = 1, zero = 0;
	if (can_reduce)
	{
		hipdnnReduceTensorDescriptor_t reduce_mean = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
		hipdnnSetReduceTensorDescriptor(reduce_mean, HIPDNN_REDUCE_TENSOR_AVG, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
		void* workspace = 0;
		size_t workspace_size = 0;
		CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce_mean, a.descriptor, b.descriptor, &workspace_size));
		if (workspace_size)
		{
			workspace = ccv_nnc_stream_context_get_workspace(stream_context, workspace_size, CCV_TENSOR_GPU_MEMORY);
			assert(workspace);
		}
		CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce_mean, 0, 0, workspace, workspace_size, &one, a.descriptor, a.data.u8, &zero, b.descriptor, b.data.u8));
		ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce_mean);
	} else if (a.data.u8 != b.data.u8) {
		// Don't need to reduce, just transfer to b, if the pointer doesn't match.
		CUDNN_ENFORCE(cudnnTransformTensor(cudnn, &one, a.descriptor, a.data.u8,  &zero, b.descriptor, b.data.u8));
	}
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_reduce_mean_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	ccv_nnc_tensor_view_t* const atv = (ccv_nnc_tensor_view_t*)outputs[0];
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, atv);
	if (inputs[0] == 0)
	{
		int i;
		ssize_t dims = 1;
		for (i = 0; i < cmd.info.reduce.count; i++)
			dims *= outputs[0]->info.dim[cmd.info.reduce.axis[i]];
		const float scale = 1.0 / (float)dims;
		CUDNN_ENFORCE(hipdnnSetTensor(cudnn, a.descriptor, a.data.u8, &scale));
	} else {
		static const float zero = 0;
		ccv_nnc_tensor_view_t gtv = ccv_nnc_get_tensor_view(inputs[0]);
		ccv_nnc_tensor_view_t* tvs[] = {
			atv, &gtv
		};
		ccv_nnc_tensor_view_alignment(tvs, 2);
		const ccv_nnc_cudnn_tensor_view_descriptor_t g = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, &gtv);
		const float scale = (float)ccv_nnc_tensor_count(inputs[0]->info) / (float)ccv_nnc_tensor_count(outputs[0]->info);
		CUDNN_ENFORCE(hipdnnAddTensor(cudnn, &scale, g.descriptor, g.data.u8, &zero, a.descriptor, a.data.u8));
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(g);
	}
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_REDUCE_MEAN_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_reduce_mean_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_REDUCE_MEAN_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_reduce_mean_back;
#endif
}

