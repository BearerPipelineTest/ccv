#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDA

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_sgd_nesterov_kernel(const size_t tensor_count, const float rate, const float decay, const float scale, const float momentum, const NUM1* const g, const NUM2* const a, const NUM2* const mom, NUM2* const b, NUM2* const new_mom)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		float grad = scale * (float)g[i];
		const float m = momentum * (float)mom[i] + grad + decay * (float)a[i];
		grad += momentum * m;
		b[i] = (NUM2)((float)a[i] - rate * grad);
		new_mom[i] = (NUM2)m;
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_sgd_kernel(const size_t tensor_count, const float rate, const float decay, const float scale, const float momentum, const float inv_dampening, const NUM1* const g, const NUM2* const a, const NUM2* const mom, NUM2* const b, NUM2* const new_mom)
{
	CUDA_1D_KERNEL_LOOP(i, tensor_count) {
		const float m = momentum * (float)mom[i] + inv_dampening * (scale * (float)g[i] + decay * (float)a[i]);
		b[i] = (NUM2)((float)a[i] - rate * m);
		new_mom[i] = (NUM2)m;
	}
}

static int _ccv_nnc_sgd_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 3);
	assert(output_size == 2);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const int nesterov = cmd.info.sgd.nesterov;
	const float rate = cmd.info.sgd.rate;
	const float scale = cmd.info.sgd.scale;
	const float decay = cmd.info.sgd.decay;
	const float momentum = cmd.info.sgd.momentum;
	const float dampening = cmd.info.sgd.dampening;
	const float inv_dampening = 1 - dampening;
	if (nesterov)
		{ assert(dampening == 0); }
	assert(inputs[1]->info.datatype == inputs[2]->info.datatype &&
		inputs[2]->info.datatype == outputs[0]->info.datatype &&
		outputs[0]->info.datatype == outputs[1]->info.datatype);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[0]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[1]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[2]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[0]));
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[1]));
	const ccv_nnc_tensor_view_t* const g = (ccv_nnc_tensor_view_t*)inputs[0];
	const ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)inputs[1];
	const ccv_nnc_tensor_view_t* const m = (ccv_nnc_tensor_view_t*)inputs[2];
	ccv_nnc_tensor_view_t* const b = (ccv_nnc_tensor_view_t*)outputs[0];
	ccv_nnc_tensor_view_t* const n = (ccv_nnc_tensor_view_t*)outputs[1];
	const size_t tensor_count = ccv_nnc_tensor_count(g->info);
	assert(tensor_count ==  ccv_nnc_tensor_count(a->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(m->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(b->info));
	assert(tensor_count ==  ccv_nnc_tensor_count(n->info));
	if (nesterov)
	{
		if (g->info.datatype == CCV_16F)
		{
			if (b->info.datatype == CCV_16F)
				_ccv_nnc_sgd_nesterov_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate, decay, scale, momentum, (__half*)g->data.f16, (__half*)a->data.f16, (__half*)m->data.f16, (__half*)b->data.f16, (__half*)n->data.f16);
			else if (b->info.datatype == CCV_32F)
				_ccv_nnc_sgd_nesterov_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate, decay, scale, momentum, (__half*)g->data.f16, a->data.f32, m->data.f32, b->data.f32, n->data.f32);
		} else if (g->info.datatype == CCV_32F) {
			if (b->info.datatype == CCV_16F)
				_ccv_nnc_sgd_nesterov_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate, decay, scale, momentum, g->data.f32, (__half*)a->data.f16, (__half*)m->data.f16, (__half*)b->data.f16, (__half*)n->data.f16);
			else if (b->info.datatype == CCV_32F)
				_ccv_nnc_sgd_nesterov_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate, decay, scale, momentum, g->data.f32, a->data.f32, m->data.f32, b->data.f32, n->data.f32);
		}
	} else {
		if (g->info.datatype == CCV_16F)
		{
			if (b->info.datatype == CCV_16F)
				_ccv_nnc_sgd_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate, decay, scale, momentum, inv_dampening, (__half*)g->data.f16, (__half*)a->data.f16, (__half*)m->data.f16, (__half*)b->data.f16, (__half*)n->data.f16);
			else if (b->info.datatype == CCV_32F)
				_ccv_nnc_sgd_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate, decay, scale, momentum, inv_dampening, (__half*)g->data.f16, a->data.f32, m->data.f32, b->data.f32, n->data.f32);
		} else if (g->info.datatype == CCV_32F) {
			if (b->info.datatype == CCV_16F)
				_ccv_nnc_sgd_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate, decay, scale, momentum, inv_dampening, g->data.f32, (__half*)a->data.f16, (__half*)m->data.f16, (__half*)b->data.f16, (__half*)n->data.f16);
			else if (b->info.datatype == CCV_32F)
				_ccv_nnc_sgd_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, rate, decay, scale, momentum, inv_dampening, g->data.f32, a->data.f32, m->data.f32, b->data.f32, n->data.f32);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_sgd_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	return CCV_NNC_EXEC_INVALID;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_SGD_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_sgd_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_SGD_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_sgd_back;
#endif
}
